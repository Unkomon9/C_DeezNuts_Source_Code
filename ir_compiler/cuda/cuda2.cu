#include <hip/hip_runtime.h>

__global__ void multiply(int* a, int* b, int* c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] * b[i];
}

int main() {
    int a[] = {1, 2, 3};
    int b[] = {1, 2, 3};
    int c[3];

    multiply<<<1, 3>>>(a, b, c);

    return 0;
}
